#include "main.cuh"

/* Wrap around CUDA RT-API calls to automatically catch and print errors */
inline
hipError_t CUDA_EC(hipError_t result)
{
#if defined(DEBUG)        
  if(result != hipSuccess) {
    fprint(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    exit(1);
  }  
#endif  
  return result;
}

int main(int argc, char ** argv)
{	
  /* Turns command line arguments into global parameters. */
	parseArgs(argc, argv);

	/* Open the source image. This is a buffer of pixels in RGBA order. */
  unsigned width, height;
	uint8_t * image = openImage(IMAGE_PATH, width, height);
  std::cout << "Opened: " << IMAGE_PATH << "\t(" << width << "x" << height << ")" << std::endl;

  /* Copy it to the GPU */
  size_t image_num_pixels = width * height;
  size_t image_size = 4 * image_num_pixels;
  uint8_t * cuda_image = 0;
  hipMallocManaged(&cuda_image, image_size);
  memcpy(cuda_image, image, image_size);

  /* Open a file to write the results */
  std::ofstream output_file;

  std::string image_path(IMAGE_PATH);
  std::string genome_length(std::to_string(GENOME_LENGTH));
  std::string pop_size(std::to_string(POPULATION_SIZE));
  std::string seed(std::to_string(RANDOM_SEED));
  std::string xover(std::to_string(XOVER_CHANCE));
  std::string mrate(std::to_string(MUTATION_RATE));

  std::string file_name = image_path;
  file_name += "_" + genome_length;
  file_name += "_" + pop_size;
  file_name += "_" + xover;
  file_name += "_" + mrate;
  file_name += "_" + seed;

  output_file.open("output/" + file_name + ".tsv");

  /* Allocate genomes on the GPU */
  size_t genome_size = GENOME_LENGTH * TRIANGLE_SIZE + BG_COLOR_SIZE; 
  size_t max_num_artists = POPULATION_SIZE + NUM_CHILDREN;

  std::vector<Artist> artists(max_num_artists);
  for(Artist & a : artists)
  {
    CUDA_EC(hipMallocManaged(&(a.genome), genome_size));
    CUDA_EC(hipMallocManaged(&(a.canvas), image_size));
  }

  /* Create a random byte generator */
  std::mt19937_64 rand_engine(RANDOM_SEED);
  std::independent_bits_engine<std::mt19937_64, 8, uint8_t> rand_byte_generator(rand_engine);

  /* Create a CUDA stream, randomize the genomes, blank out the canvases */
  for(Artist & a : artists)
  {
    /* Create the streams for each artist */
    CUDA_EC(hipStreamCreate(&(a.stream)));

    for(size_t i = 0; i < image_size; i++)
    {
      if(i < genome_size) {     
        a.genome[i] = rand_byte_generator();         
      }
    }
    /* The first bytes of a genome is the background RGBA value */
    RGBA bg_color = ((RGBA *)(a.genome))[0];
    /* Cast the canvas into a series of pixels (instead of bytes) */
    RGBA * canvas = (RGBA *)a.canvas;
    setCanvasColor<<<BAT(image_num_pixels,256)>>>(canvas, image_num_pixels, bg_color);
  }

  /* Draw the triangles to their respective canvases */
  for(Artist a : artists)
  {
    Triangle * triangles = (Triangle *)(a.genome + TRIANGLE_LIST_BEGIN);
    for(size_t i = 0; i < GENOME_LENGTH; i++)
    {
      Triangle tri = triangles[i];
      if(tri.visible == 0) { continue; }

      /* Convert the Triangle into unpacked structs for the device. */      
      Triangle_d tri_d = convertTriangleH2D(tri, width, height);
      RGBA color = convertRGBA(tri);

      drawTriangle<<<BAT(image_num_pixels,256)>>>((Pixel *)a.canvas, tri_d, color, image_num_pixels, width, height);
    }
  }
  hipDeviceSynchronize();

                    





  /* Write the image to website so I can view it */
  char artist_name[] = "artist.png";
  //saveImage(artist_name, image, width, height);
  saveImage(artist_name, artists[0].canvas, width, height);


  return 0;
}
